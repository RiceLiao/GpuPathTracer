#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optix_world.h>
#include <common.h>
#include "helpers.h"
#include "brdf_helper.cu"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float3  importance;
  int    depth;
  unsigned int seed;
};

struct PerRayData_shadow
{
    float3 visibility;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

//#define TIME_VIEW

rtBuffer<BasicLight>        lights;
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, max_depth, , );
rtDeclareVariable(unsigned int, frame_number, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void pinhole_camera()
{
  //TODO: might have issue
  const size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);
  //const float jitter_x = rnd(seed);
  //const float jitter_y = rnd(seed);
  //const float2 jitter = make_float2(jitter_x, jitter_y);
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  //const float2 d = (make_float2(launch_index) + jitter) / make_float2(launch_dim) * 2.f - 1.f;

  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  //float3 ray_direction = normalize(d.x * U + d.y * V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = make_float3(1.0f);
  prd.depth = 0;
  prd.seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);

  rtTrace(top_object, ray, prd ) ;
  output_buffer[launch_index] = make_color(prd.result);
}


RT_PROGRAM void exception()
{
  rtPrintExceptionDetails();
  output_buffer[launch_index] = make_color( bad_color );
}

rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void envmap_miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
    prd_radiance.result = make_float3(tex2D(envmap, u, v)) * prd_radiance.importance;
    //prd_radiance.result = bg_color;
}

RT_PROGRAM void closest_hit_li()
{
    float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
    float3 color = make_float3(0.0f);

    float3 hit_point = ray.origin + t_hit * ray.direction;

    for (int i = 0; i < lights.size(); ++i) {
        BasicLight light = lights[i];
        float3 L = normalize(light.pos - hit_point);

        //shadow ray
        PerRayData_shadow shadow_prd;
        shadow_prd.visibility = make_float3(1.0f); 
        float Ldist = length(light.pos - hit_point);
        optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist);
        rtTrace(top_shadower, shadow_ray, shadow_prd);
        float3 light_visibility = shadow_prd.visibility;
        float3 wi = normalize(light.pos - hit_point);
        //direct lighting
        if (fmaxf(light_visibility) > 0.0f) {
            float falloff_factor = 1.0f / (Ldist * Ldist);
            float3 Li = light.intensity_multiplier * falloff_factor * light.color;
            color += prd_radiance.importance * Li * linearblend_reflectivity_f(wi, -ray.direction, ffnormal) * max(0.0f, dot(wi, ffnormal));
        }
        //emissive lighting
        color += prd_radiance.importance * Ke * Kd;
        //color += Ke;
        //color += make_float3(Pr);

        float pdf = 1.0f;
        wi = make_float3(0.0f);
        float3 brdf = linearblend_samplewi(prd_radiance.seed, wi, -ray.direction, ffnormal, pdf);
        float cosine_term = abs(dot(wi, ffnormal));
        if (pdf < scene_epsilon){
            prd_radiance.result = color;
            return;
        }
        float3 importance = prd_radiance.importance * (brdf * cosine_term) / pdf;
        if (importance.x == 0 &&
            importance.y == 0 &&
            importance.z == 0){
            prd_radiance.result = color;
            return;
        }
        Ray reflection_ray;
        if (prd_radiance.depth < max_depth) {
            if (dot(wi, world_geo_normal) < 0) {
                reflection_ray = make_Ray(hit_point - scene_epsilon * world_geo_normal, wi, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
            }
            else {
                reflection_ray = make_Ray(hit_point + scene_epsilon * world_geo_normal, wi, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
            }

            PerRayData_radiance reflection_prd;
            reflection_prd.importance = importance;
            reflection_prd.result = color;
            reflection_prd.depth = prd_radiance.depth + 1;
            reflection_prd.seed = prd_radiance.seed;

            rtTrace(top_object, reflection_ray, reflection_prd);
            color += reflection_prd.result;
        }

    }
    prd_radiance.result = color;
}

RT_PROGRAM void shadow()
{
    prd_shadow.visibility = make_float3(0);
	rtTerminateRay();
}