#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optix_world.h>
#include <common.h>
#include "helpers.h"
#include "brdf_helper.cu"

using namespace optix;

struct PerRayData_radiance
{
  float3  result;
  float3  radiance;
  float3  importance;
  float3  origin;
  float3  direction;
  float3  albedo;
  float3  normal;
  int    depth;
  unsigned int seed;
  int  done;
};

struct PerRayData_shadow
{
    float3 visibility;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(Matrix3x3, normal_matrix, , );//TODO:

rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>              input_albedo_buffer;
rtBuffer<float4, 2>              input_normal_buffer;
rtBuffer<float4, 2>              accum_buffer;

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

//#define TIME_VIEW
rtBuffer<BasicLight>        lights;
rtBuffer<ParallelogramLight>        plights;
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, max_depth, , );
rtDeclareVariable(unsigned int, frame_number, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void pinhole_camera()
{
  //TODO: might have issue
  const size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);
  //const float jitter_x = rnd(seed);
  //const float jitter_y = rnd(seed);
  //const float2 jitter = make_float2(jitter_x, jitter_y);
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  //const float2 d = (make_float2(launch_index) + jitter) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x * U + d.y * V + W);

#if DENOISER_TYPE == 0
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
  PerRayData_radiance prd;
  prd.importance = make_float3(1.0f);
  prd.depth = 0;
  prd.seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);
  rtTrace(top_object, ray, prd ) ;
  output_buffer[launch_index] = make_float4(prd.result);
  if (prd.result.x > 1.0f || prd.result.y > 1.0f || prd.result.z > 1.0f)
  {
      output_buffer[launch_index] = make_float4(1.0f);
  }
#elif DENOISER_TYPE == 1
  float3 result = make_float3(0.0f);
  int number_of_samples = NUMBER_OF_SAMPLES;
  for (int i = 0; i < number_of_samples; i++)
  {
      optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
      PerRayData_radiance prd;
      prd.importance = make_float3(1.0f);
      prd.depth = 0;
      prd.seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1 + i);
      rtTrace(top_object, ray, prd);
      result = result + prd.result;
  }
  result = result / number_of_samples;
  output_buffer[launch_index] = make_float4(result);
#elif DENOISER_TYPE == 2
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
  PerRayData_radiance prd;
  prd.importance = make_float3(1.0f);
  prd.depth = 0;
  prd.seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);
  rtTrace(top_object, ray, prd);

  float3 result = prd.result;
  float3 albedo = prd.albedo;
  float3 normal = (length(prd.normal) > 0.f) ? normalize(normal_matrix * prd.normal) : make_float3(0., 0., 1.);

  if (frame_number > 1)
  {
      float a = 1.0f / (float)frame_number;
      float3 old_result = make_float3(output_buffer[launch_index]);
      float3 old_albedo = make_float3(input_albedo_buffer[launch_index]);
      float3 old_normal = make_float3(input_normal_buffer[launch_index]);
      output_buffer[launch_index] = make_float4(lerp(old_result, result, a));
      input_albedo_buffer[launch_index] = make_float4(lerp(old_albedo, albedo, a), 1.0f);

      float3 accum_normal = lerp(old_normal, normal, a);
      input_normal_buffer[launch_index] = make_float4((length(accum_normal) > 0.f) ? normalize(accum_normal) : normal, 1.0f);
  }
  else
  {
      output_buffer[launch_index] = make_float4(result);
      input_albedo_buffer[launch_index] = make_float4(albedo, 1.0f);
      input_normal_buffer[launch_index] = make_float4(normal, 1.0f);
  }
  output_buffer[launch_index] = make_float4(prd.result);
#elif DENOISER_TYPE == 3
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
  PerRayData_radiance prd;
  prd.importance = make_float3(1.0f);
  prd.depth = 0;
  prd.seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number + 1);
  rtTrace(top_object, ray, prd);
  float4 accum_result = make_float4(0.0f);
  if (frame_number == 1) {
      accum_result = make_float4(prd.result);
  }
  else {
      accum_result = accum_buffer[launch_index] + make_float4(prd.result);
  }
  accum_buffer[launch_index] = accum_result;
  output_buffer[launch_index] = accum_result / frame_number;
#endif
}

//RT_PROGRAM void pinhole_camera()
//{
//    int rr_begin_depth = 1;
//    unsigned int sqrt_num_samples = 2;
//
//    size_t2 screen = output_buffer.size();
//
//    float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
//    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;
//
//    float2 jitter_scale = inv_screen / sqrt_num_samples;
//    unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;
//    float3 result = make_float3(0.0f);
//
//    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number);
//    do
//    {
//        // Sample pixel using jittering
//        unsigned int x = samples_per_pixel % sqrt_num_samples;
//        unsigned int y = samples_per_pixel / sqrt_num_samples;
//        float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
//        float2 d = pixel + jitter * jitter_scale;
//        float3 ray_origin = eye;
//        float3 ray_direction = normalize(d.x * U + d.y * V + W);
//
//        // Initialze per-ray data
//        PerRayData_radiance prd;
//        prd.result = make_float3(0.f);
//        prd.importance = make_float3(1.f);
//        prd.seed = seed;
//        prd.depth = 0;
//        prd.done = false;
//
//        for (int i = 0; i < max_depth; i++)
//        //for (; ; )
//        {
//            Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
//            rtTrace(top_object, ray, prd);
//
//            if (prd.done)
//            {
//                // We have hit the background
//                prd.result += prd.radiance * prd.importance;
//                break;
//            }
//
//            // Russian roulette termination 
//            if (prd.depth >= rr_begin_depth)
//            {
//                float pcont = fmaxf(prd.importance);
//                if (rnd(prd.seed) >= pcont)
//                    break;
//                prd.importance /= pcont;
//            }
//
//            prd.depth++;
//            prd.result += prd.radiance * prd.importance;
//
//            // Update ray data for the next path segment
//            ray_origin = prd.origin;
//            ray_direction = prd.direction;
//        }
//
//        result += prd.result;
//        seed = prd.seed;
//    } while (--samples_per_pixel);
//
//    // Update the output buffer
//    float3 pixel_color = result / (sqrt_num_samples * sqrt_num_samples);
//
//    if (frame_number > 1)
//    {
//        float a = 1.0f / (float)frame_number;
//        float3 old_color = make_float3(output_buffer[launch_index]);
//        output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);
//    }
//    else
//    {
//        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
//    }
//}


RT_PROGRAM void exception()
{
  rtPrintExceptionDetails();
  output_buffer[launch_index] = make_float4(bad_color);
}

rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
    prd_radiance.result = make_float3(tex2D(envmap, u, v)) * prd_radiance.importance;
    prd_radiance.done = true;
#if DENOISER_TYPE == 2
    if (prd_radiance.depth == 0)
    {
        prd_radiance.albedo = make_float3(0.0f);
        prd_radiance.normal = make_float3(0.0f);
    }
#endif
}

RT_PROGRAM void closest_hit_li()
{
    float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
    float3 color = make_float3(0.0f);
    float3 hit_point = ray.origin + t_hit * ray.direction;

    float Ldist = 0.f;
    float3 wi = make_float3(0.0f);
    float3 L = make_float3(0.0f);
#if POINT_LIGHT
    BasicLight light = lights[0];
    float3 L = normalize(light.pos - hit_point);
    //shadow ray
    PerRayData_shadow shadow_prd;
    shadow_prd.visibility = make_float3(1.0f);
    Ldist = length(light.pos - hit_point);
    optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
    rtTrace(top_shadower, shadow_ray, shadow_prd);
    float3 light_visibility = shadow_prd.visibility;
    wi = normalize(light.pos - hit_point);

    //direct lighting
    if (fmaxf(light_visibility) > 0.0f) {
        float falloff_factor = 1.0f / (Ldist * Ldist);
        float3 Li = light.intensity_multiplier * falloff_factor * light.color;
        color += prd_radiance.importance * Li * linearblend_reflectivity_f(wi, -ray.direction, ffnormal) * max(0.0f, dot(wi, ffnormal));
    }
#endif
#if AREA_LIGHT
    // Choose random point on ParallelogramLight
    ParallelogramLight plight = plights[0];
    float z1 = rnd(prd_radiance.seed);
    float z2 = rnd(prd_radiance.seed);
    float3 light_pos = plight.corner + plight.v1 * z1 + plight.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    Ldist = length(light_pos - hit_point);
    L = normalize(light_pos - hit_point);
    float  nDl = dot(ffnormal, L);
    float  LnDl = dot(plight.normal, L);
    wi = normalize(light_pos - hit_point);

    //shadow ray
    if (nDl > 0.0f && LnDl > 0.0f)
    {
        PerRayData_shadow shadow_prd;
        shadow_prd.visibility = make_float3(1.0f);
        optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);
        float3 light_visibility = shadow_prd.visibility;

        if (fmaxf(light_visibility) > 0.0f)
        {
            const float A = length(cross(plight.v1, plight.v2));
            const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
            color += plight.emission * weight * linearblend_reflectivity_f(wi, -ray.direction, ffnormal);
        }
    }
#endif

    //emissive lighting
    color += prd_radiance.importance * Ke * Kd;

    //indirect lighting
    float pdf = 1.0f;
    float3 brdf = linearblend_samplewi(prd_radiance.seed, wi, -ray.direction, ffnormal, pdf);
    float cosine_term = abs(dot(wi, ffnormal));
    if (pdf < scene_epsilon){
        prd_radiance.result = color;
        return;
    }
    float3 importance = prd_radiance.importance * (brdf * cosine_term) / pdf;
    if (importance.x == 0 &&
        importance.y == 0 &&
        importance.z == 0){
        prd_radiance.result = color;
        return;
    }
#if DENOISER_TYPE == 2
    if (prd_radiance.depth == 0)
    {
        prd_radiance.albedo = Kd;
        prd_radiance.normal = ffnormal;
    }
#endif   
    // Russian roulette termination 
    if (prd_radiance.depth >= 1)
    {
        float pcont = fmaxf(prd_radiance.importance);
        if (rnd(prd_radiance.seed) >= pcont)
        {
            prd_radiance.result = color;
            return;
        }
    }
    if (prd_radiance.depth < max_depth) {
        //float3 R = reflect(ray.direction, ffnormal);
        Ray reflection_ray = make_Ray(hit_point - scene_epsilon * ray.direction, wi, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

        PerRayData_radiance reflection_prd;
        reflection_prd.importance = importance;
        reflection_prd.result = color;
        reflection_prd.depth = prd_radiance.depth + 1;
        reflection_prd.seed = prd_radiance.seed;
        rtTrace(top_object, reflection_ray, reflection_prd);
        color += reflection_prd.result;
    }

    prd_radiance.result = color;
}

RT_PROGRAM void closest_hit_li_cook()
{
    float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
    float3 color = make_float3(0.0f);
    float3 hit_point = ray.origin + t_hit * ray.direction;

    for (int i = 0; i < lights.size(); ++i) {
        BasicLight light = lights[i];
        float3 L = normalize(light.pos - hit_point);

        //shadow ray
        PerRayData_shadow shadow_prd;
        shadow_prd.visibility = make_float3(1.0f);
        float Ldist = length(light.pos - hit_point);
        optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);
        float3 light_visibility = shadow_prd.visibility;
        float3 wi = normalize(light.pos - hit_point);

        //direct lighting
        if (fmaxf(light_visibility) > 0.0f) {
            float falloff_factor = 1.0f / (Ldist * Ldist);
            float3 Li = light.intensity_multiplier * falloff_factor * light.color;
            color += prd_radiance.importance * Li * linearblend_reflectivity_f(wi, -ray.direction, ffnormal, 1.f) * max(0.0f, dot(wi, ffnormal));
        }

        //emissive lighting
        color += prd_radiance.importance * Ke * Kd;
    }

    prd_radiance.result = color;
}

RT_PROGRAM void closest_hit_li_ggx()
{
    float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
    float3 color = make_float3(0.0f);
    float3 hit_point = ray.origin + t_hit * ray.direction;

    for (int i = 0; i < lights.size(); ++i) {
        BasicLight light = lights[i];
        float3 L = normalize(light.pos - hit_point);

        //shadow ray
        PerRayData_shadow shadow_prd;
        shadow_prd.visibility = make_float3(1.0f);
        float Ldist = length(light.pos - hit_point);
        optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);
        float3 light_visibility = shadow_prd.visibility;
        float3 wi = normalize(light.pos - hit_point);

        //direct lighting
        if (fmaxf(light_visibility) > 0.0f) {
            float falloff_factor = 1.0f / (Ldist * Ldist);
            float3 Li = light.intensity_multiplier * falloff_factor * light.color;
            color += prd_radiance.importance * Li * linearblend_reflectivity_f(wi, -ray.direction, ffnormal, 2.f) * max(0.0f, dot(wi, ffnormal));
        }

        //emissive lighting
        color += prd_radiance.importance * Ke * Kd;
    }

    prd_radiance.result = color;
}

RT_PROGRAM void closest_hit_li_beckmann()
{
    float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
    float3 color = make_float3(0.0f);
    float3 hit_point = ray.origin + t_hit * ray.direction;

    for (int i = 0; i < lights.size(); ++i) {
        BasicLight light = lights[i];
        float3 L = normalize(light.pos - hit_point);

        //shadow ray
        PerRayData_shadow shadow_prd;
        shadow_prd.visibility = make_float3(1.0f);
        float Ldist = length(light.pos - hit_point);
        optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);
        float3 light_visibility = shadow_prd.visibility;
        float3 wi = normalize(light.pos - hit_point);

        //direct lighting
        if (fmaxf(light_visibility) > 0.0f) {
            float falloff_factor = 1.0f / (Ldist * Ldist);
            float3 Li = light.intensity_multiplier * falloff_factor * light.color;
            color += prd_radiance.importance * Li * linearblend_reflectivity_f(wi, -ray.direction, ffnormal, 3.f) * max(0.0f, dot(wi, ffnormal));
        }

        //emissive lighting
        color += prd_radiance.importance * Ke * Kd;
    }

    prd_radiance.result = color;
}

//rtTextureSampler<float4, 2> box_texture;
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
RT_PROGRAM void closest_hit_other()
{
    return;
}

RT_PROGRAM void shadow()
{
    prd_shadow.visibility = make_float3(0.0f);
	rtTerminateRay();
}